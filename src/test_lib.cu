#include "hip/hip_runtime.h"
#include <iostream>
#include <map>
#include <set>

#include "external_dependency.h"

using namespace std;

#define BLOCK_SIZE 16

__device__ double get_intersections(int* intr, int t1, int t2, int wI) {
	double n = 0;
	for (int i = 0; i < wI; i++) {
		int x1 = t1 * i;
		if (intr[x1] == 0)
			break;
		
		for (int j = 0; j < wI; j++) {
			int x2 = t2 * j;
			if (intr[x2] == 0)
				break;			
			
			if (intr[x1] == intr[x2])
				n++;
		}
	}

	return n;
}

__global__ void calc(double* result, int* tokens, int* intr, int wT, int wK, int wI) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	double t1 = (double) tokens[i];
	double t2 = (double) tokens[j];
	float v = 0;
	if (i >= j) {
		double t01 = (1 - t1) / wK;
		double t00 = -t1 / wK;
		if (i == j) {
			// calculate diagonal
			v = ((t01 * t01 * t1) + (t00 * t00 * (wK - t1))) / wK;
		} else {
			double nn = get_intersections(intr, i, j, wI);
			double t10 = -t2 / wK;
			double t11 = (1 - t2) / wK;
			v = ((nn * t01 * t11) + ((t1 - nn) * t01 * t10) + ((t2 - nn) * t00 * t11) + ((wK - (t2 + t1 - nn)) * t00 * t10)) / wK;
			//v=nn;
		}
		result[i + (j * wT)] = v;
		result[j + (i * wT)] = v;
	}
}

int covariance(map<string, int> tokens, map<string, set<int> > intersections, int wK) {
	hipFree(0);
	CHECK_CUDA_ERROR();

	int wT = tokens.size();
	Size mem_size_T = sizeof(int) * wT;
	int* h_Tokens = (int*) malloc(mem_size_T);

	// map token info to c array and copy to device
	int index = 0; // temp counter
	for (std::map<std::string, int>::iterator it = tokens.begin(); it != tokens.end(); it++) {
		h_Tokens[index++] = (*it).second;
	}

	int* d_Tokens;
	hipMalloc((void**) &d_Tokens, mem_size_T);
	hipMemcpy(d_Tokens, h_Tokens, mem_size_T, hipMemcpyHostToDevice);
	CHECK_CUDA_ERROR();

	// map intersections info to c array
	int wI = 0;
	for (map<string, set<int> >::iterator it = intersections.begin(); it != intersections.end(); it++) {
		int s = ((*it).second).size();
		if (s > wI)
			wI = s;
	}

	Size mem_size_I = sizeof(int) * wT * wI;
	int* h_Intr = (int*) malloc(mem_size_I);
	for (map<string, set<int> >::iterator it = intersections.begin(); it != intersections.end(); it++) {
		set<int> tokenSet = (*it).second;
		for (set<int>::iterator itt = tokenSet.begin(); itt != tokenSet.end(); itt++) {
			h_Intr[index++] = *itt;
		}
		// pad with zeros
		if (tokenSet.size() < wI) {
			for (int i = 0; i < wI - tokenSet.size(); i++)
				h_Intr[index++] = 0;
		}
	}

	int* d_Intr;
	hipMalloc((void**) &d_Intr, mem_size_I);
	hipMemcpy(d_Intr, h_Intr, mem_size_I, hipMemcpyHostToDevice);
	CHECK_CUDA_ERROR();

	// allocate memory for the result
	//free(*product);
	Size mem_size_result = sizeof(double) * wT * wT;
	double* h_result = (double*) malloc(mem_size_result);
	double* d_result;
	memset(h_result, 0, mem_size_result);
	hipMalloc((void **) &d_result, mem_size_result);
	hipMemset(d_result, 0, mem_size_result);

	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 numBlocks(wT / threadsPerBlock.x, wT / threadsPerBlock.y);

	calc<<<numBlocks, threadsPerBlock>>>(d_result, d_Tokens, d_Intr, wT, wK, wI);
	hipDeviceSynchronize();

	hipMemcpy(h_result, d_result, mem_size_result, hipMemcpyDeviceToHost);
	CHECK_CUDA_ERROR();

	for (int i = 0; i < (wT * wT); i++) {
		printf("%u %f\n", i, h_result[i]);
	}

	hipFree(d_Tokens);
	hipFree(d_Intr);
	hipFree(d_result);
	//delete[] h_Tokens;
	//delete[] h_Intr;
	//delete[] h_result;

	return 0;
}

