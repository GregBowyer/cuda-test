#include "hip/hip_runtime.h"
#include <iostream>
#include <map>
#include <set>

#include "external_dependency.h"

using namespace std;

#define BLOCK_SIZE 16

//#define Real double
#define Real float

__device__ Real get_intersections(int* intr, int t1, int t2, int wI) {
	int n = 0;
	for (int i = 0; i < wI; i++) {
		int x1 = (t1 * wI) + i;
		if (intr[x1] == 0)
			break;
		
		for (int j = 0; j < wI; j++) {
			int x2 = (t2 * wI) + j;
			if (intr[x2] == 0)
				break;			
			
			if (intr[x1] == intr[x2])
				n++;
		}
	}

	return (Real) n;
}

__global__ void calc(Real* result, int* tokens, int* intr, int wT, int wK, int wI) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	Real t1 = (Real) tokens[i];
	Real t2 = (Real) tokens[j];
	float v = 0;
	if (i >= j) {
		Real t00 = -t1 / wK;
		Real t01 = 1 - t1 / wK;
		if (i == j) {
			// calculate diagonal
			v = ((t01 * t01 * t1) + (t00 * t00 * (wK - t1))) / wK;
		} else {
			Real nn = get_intersections(intr, i, j, wI);
			Real t10 = -t2 / wK;
			Real t11 = 1 - t2 / wK;
			v = ((nn * t01 * t11) + ((t1 - nn) * t01 * t10) + ((t2 - nn) * t00 * t11) + ((wK - (t2 + t1 - nn)) * t00 * t10)) / wK;
		}
		result[i + (j * wT)] = v;
		result[j + (i * wT)] = v;
	}
}

int covariance(map<string, int> tokens, map<string, set<int> > intersections, int wK) {
	hipFree(0);
	CHECK_CUDA_ERROR();

	int wT = tokens.size();
	Size mem_size_T = sizeof(int) * wT;
	int* h_Tokens = (int*) malloc(mem_size_T);

	// map token info to c array and copy to device
	int index = 0; // temp counter
	for (std::map<std::string, int>::iterator it = tokens.begin(); it != tokens.end(); it++) {
		h_Tokens[index++] = (*it).second;
	}

	int* d_Tokens;
	hipMalloc((void**) &d_Tokens, mem_size_T);
	hipMemcpy(d_Tokens, h_Tokens, mem_size_T, hipMemcpyHostToDevice);
	CHECK_CUDA_ERROR();

	// map intersections info to c array
	int wI = 0;
	for (map<string, set<int> >::iterator it = intersections.begin(); it != intersections.end(); it++) {
		int s = ((*it).second).size();
		if (s > wI)
			wI = s;
	}

	index = 0;
	Size mem_size_I = sizeof(int) * wT * wI;
	int* h_Intr = (int*) malloc(mem_size_I);
	for (map<string, set<int> >::iterator it = intersections.begin(); it != intersections.end(); it++) {
		set<int> tokenSet = (*it).second;
		for (set<int>::iterator itt = tokenSet.begin(); itt != tokenSet.end(); itt++) {
			h_Intr[index++] = *itt;
		}
		// pad with zeros
		if (tokenSet.size() < wI) {
			for (int i = 0; i < wI - tokenSet.size(); i++)
				h_Intr[index++] = 0;
		}
	}
	
	int* d_Intr;
	hipMalloc((void**) &d_Intr, mem_size_I);
	hipMemcpy(d_Intr, h_Intr, mem_size_I, hipMemcpyHostToDevice);
	CHECK_CUDA_ERROR();

	// allocate memory for the result
	Size mem_size_result = sizeof(Real) * wT * wT;
	Real* h_result = (Real*) malloc(mem_size_result);
	Real* d_result;
	memset(h_result, 0, mem_size_result);
	hipMalloc((void **) &d_result, mem_size_result);
	hipMemset(d_result, 0, mem_size_result);

	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 numBlocks(wT / threadsPerBlock.x, wT / threadsPerBlock.y);

	calc<<<numBlocks, threadsPerBlock>>>(d_result, d_Tokens, d_Intr, wT, wK, wI);
	hipDeviceSynchronize();

	hipMemcpy(h_result, d_result, mem_size_result, hipMemcpyDeviceToHost);
	CHECK_CUDA_ERROR();
	
	for (int i = 0; i < (wT * wT); i++) {
		printf("%u %f\n", i, h_result[i]);
	}

	hipFree(d_Tokens);
	hipFree(d_Intr);
	hipFree(d_result);
	//delete[] h_Tokens;
	//delete[] h_Intr;
	//delete[] h_result;

	return 0;
}

