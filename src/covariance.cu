#include <iostream>
#include <map>
#include <set>

#include "covariance.h"
#include "covariance_kernel.cu"
#include "timer.h"

using namespace std;

#define BLOCK_SIZE 16

void covariance(float* h_result, map<string, int> tokens, map<string, set<int> > intersections, int wK) {
	CUDA_SAFE_CALL(hipFree(0));
	CUTimer *mem_timer = start_timing("Memory handling time");

	int wT = tokens.size();
	Size mem_size_T = sizeof(int) * wT;
	int* h_Tokens = (int*) malloc(mem_size_T);

	// map token info to c array and copy to device
	int index = 0; // temp counter
	for (std::map<std::string, int>::iterator it = tokens.begin(); it != tokens.end(); it++) {
		h_Tokens[index++] = (*it).second;
	}

	int* d_Tokens;
	CUDA_SAFE_CALL(hipMalloc((void**) &d_Tokens, mem_size_T));
	CUDA_SAFE_CALL(hipMemcpy(d_Tokens, h_Tokens, mem_size_T, hipMemcpyHostToDevice));

	// map intersections info to c array
	int wI = 0;
	for (map<string, set<int> >::iterator it = intersections.begin(); it != intersections.end(); it++) {
		int s = ((*it).second).size();
		if (s > wI)
			wI = s;
	}

	index = 0;
	Size mem_size_I = sizeof(int) * wT * wI;
	int* h_Intr = (int*) malloc(mem_size_I);
	for (map<string, set<int> >::iterator it = intersections.begin(); it != intersections.end(); it++) {
		set<int> tokenSet = (*it).second;
		for (set<int>::iterator itt = tokenSet.begin(); itt != tokenSet.end(); itt++) {
			h_Intr[index++] = *itt;
		}
		// pad with zeros
		if (tokenSet.size() < wI) {
			for (int i = 0; i < wI - tokenSet.size(); i++)
				h_Intr[index++] = 0;
		}
	}
	
	int* d_Intr;
	CUDA_SAFE_CALL(hipMalloc((void**) &d_Intr, mem_size_I));
	CUDA_SAFE_CALL(hipMemcpy(d_Intr, h_Intr, mem_size_I, hipMemcpyHostToDevice));

	// allocate memory for the result
	Size mem_size_result = sizeof(float) * wT * wT;
	//float* h_result = (float*) malloc(mem_size_result);
	float* d_result;
	//memset(h_result, 0, mem_size_result);
	CUDA_SAFE_CALL(hipMalloc((void **) &d_result, mem_size_result));
	CUDA_SAFE_CALL(hipMemset(d_result, 0, mem_size_result));

	finish_timing(mem_timer);

	CUTimer *calculation_timer = start_timing("Calculation on card");
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 numBlocks(wT / threadsPerBlock.x, wT / threadsPerBlock.y);

	calc<<<numBlocks, threadsPerBlock>>>(d_result, d_Tokens, d_Intr, wT, wK, wI);
	hipDeviceSynchronize();
	CUDA_CHECK_ERROR();

	CUDA_SAFE_CALL(hipMemcpy(h_result, d_result, mem_size_result, hipMemcpyDeviceToHost));
	
	hipFree(d_Tokens);
	hipFree(d_Intr);
	hipFree(d_result);
	delete h_Tokens;
	delete h_Intr;
	finish_timing(calculation_timer);
}

