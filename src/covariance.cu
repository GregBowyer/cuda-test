#include <iostream>
#include <map>
#include <set>

#include "covariance.h"
#include "covariance_kernel.cu"
#include "timer.h"

using namespace std;

#define BLOCK_SIZE 16

void covariance(float* h_result, map<string, int> tokens, map<string, set<int> > intersections, int wK) {
	Cuda_SAFE_CALL(hipFree(0));
	CUTimer *gpu_total_timer = start_timing("[GPU] Total Time");
	CUTimer *mem_timer = start_timing("[GPU] Host→Device Memory Load");

	int wT = tokens.size();
	size_t mem_size_T = sizeof(int) * wT;
	int* h_Tokens = (int*) malloc(mem_size_T);

	// map token info to c array and copy to device
	int index = 0; // temp counter
	for (std::map<std::string, int>::iterator it = tokens.begin(); it != tokens.end(); it++) {
		h_Tokens[index++] = (*it).second;
	}

	int* d_Tokens;
	Cuda_SAFE_CALL(hipMalloc((void**) &d_Tokens, mem_size_T));
	Cuda_SAFE_CALL(hipMemcpy(d_Tokens, h_Tokens, mem_size_T, hipMemcpyHostToDevice));

	// map intersections info to c array
	int wI = 0;
	for (map<string, set<int> >::iterator it = intersections.begin(); it != intersections.end(); it++) {
		int s = ((*it).second).size();
		if (s > wI)
			wI = s;
	}

	index = 0;
	size_t mem_size_I = sizeof(int) * wT * wI;
	int* h_Intr = (int*) malloc(mem_size_I);
	for (map<string, set<int> >::iterator it = intersections.begin(); it != intersections.end(); it++) {
		set<int> tokenSet = (*it).second;
		for (set<int>::iterator itt = tokenSet.begin(); itt != tokenSet.end(); itt++) {
			h_Intr[index++] = *itt;
		}
		// pad with zeros
		if (tokenSet.size() < wI) {
			for (int i = 0; i < wI - tokenSet.size(); i++)
				h_Intr[index++] = -1;
		}
	}
	
	int* d_Intr;
	Cuda_SAFE_CALL(hipMalloc((void**) &d_Intr, mem_size_I));
	Cuda_SAFE_CALL(hipMemcpy(d_Intr, h_Intr, mem_size_I, hipMemcpyHostToDevice));

	// allocate memory for the result
	size_t mem_size_result = sizeof(float) * wT * wT;
	//float* h_result = (float*) malloc(mem_size_result);
	float* d_result;
	//memset(h_result, 0, mem_size_result);
	Cuda_SAFE_CALL(hipMalloc((void **) &d_result, mem_size_result));
	Cuda_SAFE_CALL(hipMemset(d_result, 0, mem_size_result));

	finish_timing(mem_timer);

	CUTimer *calculation_timer = start_timing("[GPU] Calculation on card");
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 numBlocks(wT / threadsPerBlock.x, wT / threadsPerBlock.y);

	calc<<<numBlocks, threadsPerBlock>>>(d_result, d_Tokens, d_Intr, wT, wK, wI);
	hipDeviceSynchronize();
	Cuda_CHECK_ERROR();
	finish_timing(calculation_timer);

	CUTimer *backcopy_timer = start_timing("[GPU] Device→Host Memory Fetch");
	Cuda_SAFE_CALL(hipMemcpy(h_result, d_result, mem_size_result, hipMemcpyDeviceToHost));
	
	hipFree(d_Tokens);
	hipFree(d_Intr);
	hipFree(d_result);
	delete h_Tokens;
	delete h_Intr;
	finish_timing(backcopy_timer);
	finish_timing(gpu_total_timer);
}

